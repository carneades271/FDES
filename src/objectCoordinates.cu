/*==================================================================

Copyright (C) 2015 Wouter Van den Broek, Xiaoming Jiang

This file is part of FDES.

FDES is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

FDES is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with FDES. If not, see <http://www.gnu.org/licenses/>.

Email: wouter.vandenbroek@uni-ulm.de, wouter.vandenbroek1@gmail.com,
       xiaoming.jiang@uni-ulm.de, jiang.xiaoming1984@gmail.com 

===================================================================*/

#include "objectCoordinates.h"
# include "paramStructure.h"

void getCoordinates( int nAt, int** Z_d, float** xyzCoord_d, float** DWF_d, float** occ_d )
{
	FILE* fr;
	fr = fopen ( "Params.cnf", "rt" );

	cuda_assert ( hipMalloc ( ( void** ) Z_d, nAt * sizeof ( int ) ) );
	cuda_assert ( hipMalloc ( ( void** ) xyzCoord_d, nAt * 3 * sizeof ( float ) ) );
	cuda_assert ( hipMalloc ( ( void** ) DWF_d, nAt * sizeof ( float ) ) );
	cuda_assert ( hipMalloc ( ( void** ) occ_d, nAt * sizeof ( float ) ) );

	int *Z_h;
	float *xyzCoord_h, *DWF_h, *occ_h;
	Z_h = ( int* ) malloc ( nAt * sizeof ( int ) );
	xyzCoord_h = ( float* ) malloc ( nAt * 3 * sizeof ( float ) );
	DWF_h = ( float* ) malloc ( nAt * sizeof ( float ) );
	occ_h = ( float* ) malloc ( nAt * sizeof ( float ) );

	// Read values
	readCoordinates ( fr, Z_h, xyzCoord_h, DWF_h, occ_h );

	cuda_assert ( hipMemcpy ( *Z_d, Z_h,  nAt * sizeof ( int ), hipMemcpyHostToDevice ) );
	cuda_assert ( hipMemcpy ( *xyzCoord_d, xyzCoord_h, nAt * 3 * sizeof ( int ), hipMemcpyHostToDevice ) );
	cuda_assert ( hipMemcpy ( *DWF_d, DWF_h, nAt * sizeof ( float ), hipMemcpyHostToDevice ) );
	cuda_assert ( hipMemcpy ( *occ_d, occ_h, nAt * sizeof ( float ), hipMemcpyHostToDevice ) );

	fclose ( fr );
	free( Z_h );
	free( xyzCoord_h );
	free( DWF_h );
	free( occ_h );
}


void freeCoordinatesVars ( int* Z_d, float* xyzCoord_d, float* DWF_d, float* occ_d )
{
	cuda_assert ( hipFree ( Z_d ) );
	cuda_assert ( hipFree ( xyzCoord_d ) );
	cuda_assert ( hipFree ( DWF_d ) );
	cuda_assert ( hipFree ( occ_d ) );
}


void readCoordinates ( FILE* fr, int* Z_h, float* xyzCoord_h, float* DWF_h, float* occ_h )
{
	fseek ( fr, 0, SEEK_SET );
	const int size = 200;
	char* line;
	line = ( char* ) malloc ( size * sizeof ( char ) );
	char* fieldName;
	fieldName = ( char* ) malloc ( size * sizeof ( char ) );
	int j = 0;

	while ( !feof( fr ) ) {
		fgets ( line, size, fr );
		sscanf ( line, "%s", fieldName );
		if ( !strncmp ( fieldName, "atom:", 5 ) ) {    
			sscanf ( line, "%*s %i %g %g %g %g %g", & Z_h[j], & xyzCoord_h[3 * j + 0], & xyzCoord_h[3 * j + 1], & xyzCoord_h[3 * j + 2], & DWF_h[j], & occ_h[j] );
			j += 1;
		}
		resetLine ( line );
	}

	free ( line );
	free ( fieldName );
}

void resetLine( char* line )
{
	line[0] = *( "#" ); // Reset "loremipsum" to "#oremipsum", indicating a comment.
}